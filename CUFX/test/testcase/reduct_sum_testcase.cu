#include "compare.cuh"
#include "test_case.cuh"
#include "op_external.cuh"
#include "matrix.cuh"

template <typename T, typename U>
T ReductSumCRun(const Matrix &src, T &val) {
    if (val != 0) {
        val = 0;
    }

    for (int h = 0; h < src.height; h++) {
        for (int w = 0; w < src.width; w++) {
            for (int c = 0; c < src.channel; c++) {
                val += src.At<U>(h, w, c);
            }
        }
    }

    return 0;
}

TestCase(CudaOp, ReductSumFloat) {
    Matrix src{ElemType::ElemFloat, {10, 40, 3}, MemoryType::GlobalMemory, IsAsync::IsAsyncFalse};
    hipError_t cuda_ret = src.MatrixCreate();

    src.GetBytes<float>();

    int c_ret = 0;
    ASSERT_EQ(cuda_ret, 0);

    float c_res = -1;
    float cuda_res = -1;

    // cuda run
    {
        cuda_ret = ReductSum(src, &cuda_res);
        ASSERT_EQ(cuda_ret, 0);
    }

    // C run
    {
        c_ret = ReductSumCRun<float, float>(src, c_res);
        ASSERT_EQ(c_ret, 0); // 运行成功应该返回 0
    }

    ASSERT_NEAREQ(c_res, cuda_res);
}

TestCase(CudaOp, ReductSumInt) {
    Matrix src{ElemType::ElemInt, {4, 4, 3}, MemoryType::GlobalMemory, IsAsync::IsAsyncFalse};
    hipError_t cuda_ret = src.MatrixCreate();

    src.GetBytes<int>();

    int c_ret = 0;
    ASSERT_EQ(cuda_ret, 0);

    ulong c_res = 1;
    ulong cuda_res = 1;

    // cuda run
    {
        cuda_ret = ReductSum(src, &cuda_res);
        ASSERT_EQ(cuda_ret, 0);
    }

    // C run
    {
        c_ret = ReductSumCRun<ulong, int>(src, c_res);
        ASSERT_EQ(c_ret, 0); // 运行成功应该返回 0
    }

    ASSERT_EQ(c_res, cuda_res);
}