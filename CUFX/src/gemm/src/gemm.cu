#include "hip/hip_runtime.h"
#include "op_external.cuh"
#include "clock.cuh"
#include "runtime_info.cuh"

template <typename T>
__global__ void GemmKernel(T *src1, T *src2, T *dst, std::size_t h, std::size_t k, std::size_t w) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int height = blockIdx.y * blockDim.y + ty;
    const int width = blockIdx.x * blockDim.x + tx;

    if (width >= w || height >= h) {
        return;
    }

    float sum = 0.0f;

    for (int i = 0; i < k; i++) {
        sum += src1[height * k + i] * src2[w * i + width];
    }

    dst[height * w + width] = sum;
}

template <typename T>
hipError_t GemmImpl(const Matrix &src1, const Matrix &src2, Matrix &dst) {
    hipError_t ret = hipSuccess;

    int src1_height = src1.height;
    int src1_width = src1.width;
    int src2_width = src2.width;

    int local_height = 32;
    int local_width = 32;

    dim3 grid_size = GetGridSize(dst.width, dst.height, local_width, local_height);
    dim3 block_size(local_width, local_height);

    ProfileTime time{"Gemm"};
    time.StartGpuTime();
    GemmKernel<T><<<grid_size, block_size>>>(src1.GetCudaData<T>(), src2.GetCudaData<T>(), dst.GetCudaData<T>(),
                                             src1_height, src1_width, src2_width);
    time.EndGpuTime();

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(dst.SyncToHost<T>());

    return ret;
}

hipError_t Gemm(const Matrix &src1, const Matrix &src2, Matrix &dst) {
    hipError_t ret = hipSuccess;

    if (src1.elem_type != ElemType::ElemFloat || src2.elem_type != ElemType::ElemFloat
        || dst.elem_type != ElemType::ElemFloat) {
        LOGE("only support float matrix for gemm now \n");
        return hipErrorInvalidValue;
    }

    ret = GemmImpl<float>(src1, src2, dst);
    return ret;
}
